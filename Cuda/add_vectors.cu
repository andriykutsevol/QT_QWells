/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */
#include "../src/header.h"
#include "../../common/book.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cula.h>


int f_cuda_add_vec( void ) {

    int c;
    int *dev_c;
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );

    add_ker<<<1,1>>>( 2, 7, dev_c );

    HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int),
                              hipMemcpyDeviceToHost ) );
    printf( "2 + 7 = %d\n", c );
    HANDLE_ERROR( hipFree( dev_c ) );

    printf( "Hello from CUDA!\n" );
    return 0;
}


void f_curand_host_api(void){

	size_t n = 10;
	size_t i;

	hiprandGenerator_t gen;

	float *devData, *hostData;

	hostData = (float *) calloc(n, sizeof(float));

	hipMalloc((void **)&devData, n*sizeof(float));

	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);

	hiprandSetPseudoRandomGeneratorSeed(gen, 123ULL);

	hiprandGenerateUniform(gen, devData, n);

	hipMemcpy(hostData, devData, n*sizeof(float), hipMemcpyDeviceToHost);

	printf("Random Unif(0,1) draws:\n");
	for(i=0; i<n; i++){
		printf("%1.4f\n", hostData[i]);
	}
	printf("\n");

	hiprandDestroyGenerator(gen);
	hipFree(devData);
	free(hostData);
}


void f_curand_dev_api(){
	const int N = 20;
	int i, total;

	int *devResults, *hostResults;
	hiprandState *devStates;

	hostResults = (int *) calloc(N*N, sizeof(int));

	hipMalloc((void **)&devResults, N*N*sizeof(int));

	hipMemset(devResults, 0, N*N*sizeof(int));

	hipMalloc((void**)&devStates, N*N*sizeof(hiprandState));

	setup_kernel<<<N,N>>>(devStates);

	//генерировать и использовать псевдослучайные числа.
	//for(i = 0; i<10; i++){
	//	std::cout << "from for " << i << std::endl;
	generate_kernel <<< N,N >>>(devStates, devResults);
	//}
	hipDeviceSynchronize();

	hipMemcpy(hostResults, devResults, N*N*sizeof(int), hipMemcpyDeviceToHost);
	// показать резы.
	total = 0;
	for (i=0; i<3*3; i++){
		printf("hostResults\n");
		total += hostResults[i];
	}
	printf("Fraction odd was%10.13f\n", (float) total /(3.0f * 3.0f * 10.0f * 10.0f));

	hipFree(devStates);
	hipFree(devResults);
	free(hostResults);

// The END;
}


void f_cula_dev(void){

	// кроме настроек в eclipse надо еще создасть ссылки
	// из /usr/local/cula/lib64/ в /usr/lib/

	culaStatus s;

	s = culaInitialize();
	if(s != culaNoError)
	{
		printf("%s\n", culaGetStatusString(s));
    // /* ... Error Handling ... */
	}

	/* ... Your code ... */

	culaShutdown();

}
















