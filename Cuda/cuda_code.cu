#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <cstdlib>

#include "common/book.h"
#include "header.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// http://stackoverflow.com/questions/13245258/handle-error-not-found-error-in-cuda
//static void HandleError( hipError_t err,
//                         const char *file,
//                         int line ) {
//    if (err != hipSuccess) {
//        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
//                file, line );
//        exit( EXIT_FAILURE );
//    }
//}
//#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void add(int a, int b, int *c){
    *c = a + b;
}

extern "C"
void cuda_main()
{
        int c;
        int *dev_c;
        HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );

        add<<<1,1>>>( 2, 7, dev_c );

        HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int),
                                  hipMemcpyDeviceToHost ) );
        printf( "2 + 7 = %d\n", c );
        HANDLE_ERROR( hipFree( dev_c ) );


   // return hipGetLastError();
}



//====================================================================


// //*/ generate 32M random numbers on the host
//    thrust::host_vector<int> h_vec(32 << 20);
//    thrust::generate(h_vec.begin(), h_vec.end(), rand);

//    // transfer data to the device
//    thrust::device_vector<int> d_vec = h_vec;

//    // sort data on the device (846M keys per second on GeForce GTX 480)
//    thrust::sort(d_vec.begin(), d_vec.end());

//    // transfer data back to host
//    thrust::copy(d_vec.begin(), d_vec
