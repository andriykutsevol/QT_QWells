#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include "../src/header.h"

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


__global__ void add_ker( int a, int b, int *c ) {
	printf("hello from kernel\n");
    *c = a + b;
}


__global__ void setup_kernel(hiprandState *state){
	int id = threadIdx.x + blockIdx.x * blockDim.x;    // 64 = blockDim.x
	printf("KER_SET:  id, blockIdx.x, threadIdx.x = %d, %d, %d\n", id, blockIdx.x, threadIdx.x);
	hiprand_init(1, id, 0, &state[id]);   // (seed, sequence, offset, hiprandState *state)

}

__global__ void generate_kernel(hiprandState *state, int *result){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int count = 0;
	unsigned int x;
	hiprandState localState = state[id];
	x = hiprand(&localState);
	printf("KER_GEN: blockIdx.x, threadIdx.x, x = %d,%d, %d\n",blockIdx.x, threadIdx.x, x);
	// Check if odd;
	if(x & 1){
		count ++;
	}
	state[id] = localState;
	result[id] += count;
}
