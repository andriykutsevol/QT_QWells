#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include <hip/hip_vector_types.h>
#include <typeinfo>

#include <string>
#include <vector>
#include <numeric>
#include <stdexcept>


#include <cula.h>

const int N = 33*1024;
const int numSims = 400000;
// для успешной редукции threadsPerBlock должно быть степерью 2
// 2^9 = 512;
const int threadsPerBlock = 512;
const int blocksPerGrid = 16;

//==============================


__global__ void add_ker( int a, int b, int *c ) {
	printf("hello from kernel\n");
    *c = a + b;
}

int f_cuda_add_vec( int a, int b) {

    int c;
    int *dev_c;
    hipMalloc( (void**)&dev_c, sizeof(int) );

    add_ker<<<1,1>>>( a, b, dev_c );

    hipMemcpy( &c, dev_c, sizeof(int),
                              hipMemcpyDeviceToHost );
    printf( "2 + 7 = %d\n", c );
    hipFree( dev_c );

   // hiprandDirectionVectors64_t *rngDirections;
   // hiprandGetDirectionVectors64(&rngDirections, HIPRAND_DIRECTION_VECTORS_64_JOEKUO6);

    printf( "Hello from CUDA!\n" );
    return c;
}


//======================================
//http://kvm.gubkin.ru/vip3p2/g4.pdf
//======================================

// это из самой cuda.
//void hiprand_init(hiprandDirectionVectors64_t direction_vectors,
//                                           unsigned long long offset,
//                                           hiprandStateSobol64_t *state)

__global__ void setup_sobols_kernel(hiprandStateSobol64_t *d_sobolStates,
		int dim_xn, hiprandDirectionVectors64_t *d_rngDirections, int numSims){
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int step = gridDim.x * blockDim.x;

	while(tid < numSims){
		for(int ldim=0; ldim < dim_xn; ldim++){
			hiprand_init(d_rngDirections[ldim], tid, &d_sobolStates[tid + ldim*numSims]);
		}
		tid += step;
	}

}

//================================

__device__ void getPoints(double *xn, hiprandStateSobol64_t *d_sobolStates,
		double *low_lims, double *top_lims, int tid, int dim_xn, int numSims){

	for(int d=0; d<dim_xn; d++){
		xn[d] = (hiprand_uniform_double(&d_sobolStates[tid + numSims*d]) *
				(top_lims[d] - low_lims[d])) + low_lims[d];
	}
}

//================================

//__global__ void get_sums_kernel(hiprandStateSobol64_t *d_sobolStates,
//		double *d_results, int numSims){
//
//	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	unsigned int step = gridDim.x * blockDim.x;
//
//
//	const int dim_xn = 4;
//	double xn_s[dim_xn];
//	double low_lims[dim_xn];
//	double top_lims[dim_xn];
//
//	low_lims[0] = 1; low_lims[1] = 1;  low_lims[2] = 0; low_lims[3] = 0; 
//	top_lims[0] = 3; top_lims[1] = 27; top_lims[2] = 90; top_lims[3] = 15;
//
//	while(tid < numSims){
//		getPoints(&xn_s[0], d_sobolStates, &low_lims[0], &top_lims[0], tid, dim_xn, numSims);
//		if(  (xn_s[1] > xn_s[0]) && ( xn_s[1] < pow(xn_s[0],3) )     ){
//			if(   (xn_s[2] > 0) && ( xn_s[2] < (pow(xn_s[0],2) + xn_s[0] * xn_s[1]) )    ){
//				//d_results[tid] = 1;
//				if( (xn_s[3] > 0) && (xn_s[3] < (xn_s[2] / 6.0)) ) {  
//					d_results[tid] = 1;
//				}else{
//					d_results[tid] = 0;
//				}
//			}else{
//				d_results[tid] = 0;
//			}
//		}else{
//			d_results[tid] = 0;
//		}
//
//		tid += step;
//	}
//
//}

__global__ void get_sums_kernel(hiprandStateSobol64_t *d_sobolStates,
		double *d_results, int numSims){

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int step = gridDim.x * blockDim.x;

	int cacheIndex = threadIdx.x;
	__shared__ float cache[threadsPerBlock];
	// локальная копия cache создается в каждом блоке.
	// поэтому и размер cache = колличеству нитей в одном лишь блоке.

	const int dim_xn = 4;
	double xn_s[dim_xn];
	double low_lims[dim_xn];
	double top_lims[dim_xn];

	low_lims[0] = 1; low_lims[1] = 1;  low_lims[2] = 0; low_lims[3] = 0; 
	top_lims[0] = 3; top_lims[1] = 27; top_lims[2] = 90; top_lims[3] = 15;

	float temp = 0;
	// это линейное вычисление внутри самой нити.
	// т.е мы досчитываем в нити то,
	// на что не хватило размерностей сетки. Поэтому и temp не векторный.
	while(tid < numSims){
		getPoints(&xn_s[0], d_sobolStates, &low_lims[0], &top_lims[0], tid, dim_xn, numSims);
		if(  (xn_s[1] > xn_s[0]) && ( xn_s[1] < pow(xn_s[0],3) )     )
			if(   (xn_s[2] > 0) && ( xn_s[2] < (pow(xn_s[0],2) + xn_s[0] * xn_s[1]) )    )
				if( (xn_s[3] > 0) && (xn_s[3] < (xn_s[2] / 6.0)) ) 
					temp ++;

		tid += step;
	}
	// cacheIndex = threadIdx.x ;
	// теперь уже для каждой нити мы записываем результат.
	cache[cacheIndex] = temp;
	__syncthreads();
	// все нити отработали
	// теперь получим сумму для каждого блока.

	// стр.75. Каждая нить объединяет два элемента в один. По завершении
	// одного шага кол-во элементов в массиве уменьшится вдвое.
	// Потом __syncthreads(), потом опять уменьшится вдвое.
	// И так за log2(threadsPerBlock) шагов вычислим сумму
	// всех элементов массива cache[].
	int i = blockDim.x / 2;
	while(i != 0){
		if(cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if(cacheIndex == 0)
		d_results[blockIdx.x] = cache[0];

}

//===============================
// Интеграл с отрицательными значениями.
// Найти площадь плоской фигуры, ограниченной линиями
// y = 2x - x^2, y = -x;
//===============================

__global__ void get_sums_kernel_2(hiprandStateSobol64_t *d_sobolStates,
		double *d_results, int numSims){

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int step = gridDim.x * blockDim.x;

	int cacheIndex = threadIdx.x;
	__shared__ float cache[threadsPerBlock];
	// локальная копия cache создается в каждом блоке.
	// поэтому и размер cache = колличеству нитей в одном лишь блоке.

	const int dim_xn = 2;
	double xn_s[dim_xn];
	double low_lims[dim_xn];
	double top_lims[dim_xn];

	low_lims[0] = 0; low_lims[1] = 1;
	top_lims[0] = 3; top_lims[1] = -3;

	float temp = 0;
	// это линейное вычисление внутри самой нити.
	// т.е мы досчитываем в нити то,
	// на что не хватило размерностей сетки. Поэтому и temp не векторный.
	while(tid < numSims){

		getPoints(&xn_s[0], d_sobolStates, &low_lims[0], &top_lims[0], tid, dim_xn, numSims);
		if(xn_s[1] > 0){
			if((xn_s[1] > (-xn_s[0])) && (xn_s[1] < (2 * xn_s[0] - xn_s[0] * xn_s[0]))){
				temp ++;
				printf("%f %f \n", xn_s[0], xn_s[1]);
			}else{
				temp --;
			}
		}
		tid += step;
	}
	// cacheIndex = threadIdx.x ;
	// теперь уже для каждой нити мы записываем результат.
	cache[cacheIndex] = temp;
	__syncthreads();
	// все нити отработали
	// теперь получим сумму для каждого блока.

	// стр.75. Каждая нить объединяет два элемента в один. По завершении
	// одного шага кол-во элементов в массиве уменьшится вдвое.
	// Потом __syncthreads(), потом опять уменьшится вдвое.
	// И так за log2(threadsPerBlock) шагов вычислим сумму
	// всех элементов массива cache[].
	int i = blockDim.x / 2;
	while(i != 0){
		if(cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if(cacheIndex == 0)
		d_results[blockIdx.x] = cache[0];

}

//================================

//===============================
// Интеграл с отрицательными значениями.
// Вычислить именно интеграл
// int_0^3pi/2 cos(x) dx.
//===============================

__global__ void get_sums_kernel_3(hiprandStateSobol64_t *d_sobolStates,
		double *d_results, int numSims){

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int step = gridDim.x * blockDim.x;

	int cacheIndex = threadIdx.x;
	__shared__ float cache[threadsPerBlock];
	// локальная копия cache создается в каждом блоке.
	// поэтому и размер cache = колличеству нитей в одном лишь блоке.

	const int dim_xn = 2;		 
	double xn_s[dim_xn];
	double low_lims[dim_xn];
	double top_lims[dim_xn];

	low_lims[0] = 0;              low_lims[1] = -1; 
	top_lims[0] = (3 * 3.14 / 2); top_lims[1] =  1;

	float temp = 0;
	// это линейное вычисление внутри самой нити.
	// т.е мы досчитываем в нити то,
	// на ++ сетки. Поэтому и temp не векторный.
	while(tid < numSims){

		getPoints(&xn_s[0], d_sobolStates, &low_lims[0], &top_lims[0], tid, dim_xn, numSims);
		if((xn_s[1] > 0) && (xn_s[1] < (cos(xn_s[0])))) {
			//printf("%f %f \n", xn_s[0], xn_s[1]);
			temp ++;
		}else if((xn_s[1] < 0) && (xn_s[1] > cos(xn_s[0]))){
			temp --;
			//printf("%f %f \n", xn_s[0], xn_s[1]);
		}

		tid += step;
	}
	// cacheIndex = threadIdx.x ;
	// теперь уже для каждой нити мы записываем результат.
	cache[cacheIndex] = temp;
	__syncthreads();
	// все нити отработали
	// теперь получим сумму для каждого блока.

	// стр.75. Каждая нить объединяет два элемента в один. По завершении
	// одного шага кол-во элементов в массиве уменьшится вдвое.
	// Потом __syncthreads(), потом опять уменьшится вдвое.
	// И так за log2(threadsPerBlock) шагов вычислим сумму
	// всех элементов массива cache[].
	int i = blockDim.x / 2;
	while(i != 0){
		if(cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if(cacheIndex == 0)
		d_results[blockIdx.x] = cache[0];

}


int f_monte_cudev(){



	// Размерность устанавливается тут и в get_sums_kernel
	// в виде const int dim_xn = 3;
	//int dim_xn = 4;			// !! для get_sums_kernel
	int dim_xn = 2;			// !! для get_sums_kernel_2
	dim3 block;
	block.x = threadsPerBlock;
	dim3 grid;
	grid.x = blocksPerGrid;


	hiprandStateSobol64_t *d_sobolStates = 0;
	// dimmensional - dim_xn*numSims
	// состояний столько сколько симуляций * размерность.
	hipMalloc((void **)&d_sobolStates, dim_xn*numSims * sizeof(hiprandStateSobol64_t));

	hiprandDirectionVectors64_t *d_rngDirections = 0;
	// dimmensional - dim_xn
	// векторов направлений столько, сколько размерностей.
	hipMalloc((void **)&d_rngDirections, dim_xn * sizeof(hiprandDirectionVectors64_t));

	hiprandDirectionVectors64_t *rngDirections;
	hiprandGetDirectionVectors64(&rngDirections, HIPRAND_DIRECTION_VECTORS_64_JOEKUO6);

	hipMemcpy(d_rngDirections, rngDirections, dim_xn * sizeof(hiprandDirectionVectors64_t), hipMemcpyHostToDevice);

	//setup_sobol_kernel<<<grid, block>>>(d_sobolStates, d_rngDirections, numSims);
	setup_sobols_kernel<<<grid, block>>>(d_sobolStates, dim_xn, d_rngDirections, numSims);


	double *d_results;
	hipMalloc((void **)&d_results, numSims * sizeof(double));

	//get_sums_kernel<<<grid, block>>>(d_sobolStates, d_results, numSims);
	//get_sums_kernel_2<<<grid, block>>>(d_sobolStates, d_results, numSims);
	get_sums_kernel_3<<<grid, block>>>(d_sobolStates, d_results, numSims);

	hipDeviceSynchronize();

	double *host_results;
	host_results = (double*)calloc( numSims, sizeof(double) );
	hipMemcpy(host_results, d_results, numSims * sizeof(double), hipMemcpyDeviceToHost );
	double sum = 0;

	for(int k = 0; k < blocksPerGrid; k++){
		sum += host_results[k];
	}

//	for(int k=0; k<(numSims); k++){
//		// то есть учитываем только точки, которые попали в область определения.
//		sum += host_results[k];
//	}
	//std::cout << "host_sum: " << sum << "; integral is: " << (2*26*90*15* sum / numSims) << std::endl;		 
	std::cout << "host_sum: " << sum << "; integral is: " << (2*(3*3.14/2) * sum / numSims) << std::endl;		 

	return 0;
}
