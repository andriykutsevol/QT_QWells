#include "hip/hip_runtime.h"

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <math.h>


#include <hip/hip_vector_types.h>
#include <typeinfo>

#include <string>
#include <vector>
#include <numeric>


#include "phontan_cuda_lib.h"
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#include <cuda_gl_interop.h>

#include <time.h>

const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 256; //256;
const unsigned int mesh_height = 256; //256;

//andy
const unsigned int max_age = 10000;

// vbo variables
GLuint vbo;
GLfloat vert_data[] = {-1.0f, 1.0f, -1.0f,
1.0f, 1.0f, -1.0f,
1.0f, -1.0f, -1.0f};

float anim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

//andy
float * d_particleData;
float * h_particleData;

float targetX, targetY, targetZ;



///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u*freq + time) * sinf(v*freq + time) * 0.5f;

    // write output vertex
    pos[y*width+x] = make_float4(u, w, v, 1.0f);
}



void cudaUpdateBuff(struct hipGraphicsResource **vbo_resource, unsigned int vbo_res_flags,
		GLuint *vbo,
		const unsigned int mesh_width,
		const unsigned int mesh_height, float g_fAnim){

	 // register this buffer object with CUDA
	    hipGraphicsGLRegisterBuffer(vbo_resource, *vbo, vbo_res_flags);

	 // map OpenGL buffer object for writing from CUDA
	   	float4 *dptr;
	   	hipGraphicsMapResources(1, vbo_resource, 0);
	   	size_t num_bytes;
	   	hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
	   	                                                         *vbo_resource);


	   // launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);
	    // execute the kernel
	    dim3 block(8, 8, 1);
	    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	    simple_vbo_kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

	    // unmap buffer object
	    hipGraphicsUnmapResources(1, vbo_resource, 0);
	    hipGraphicsUnregisterResource(*vbo_resource);

	    //std::cout<<"end cudaUpdateBuff." << std::endl;

}












































void runCuda(GLuint vbo){
//	       glBegin(GL_TRIANGLES);
//	           glVertex3f(0.0f,1.0f,0.0f);
//	           glVertex3f(-1.0f,-1.0f,0.0f);
//	           glVertex3f(1.0f,-1.0f,0.0f);
//	       glEnd();


	float4 *dptr;

	cudaGLRegisterBufferObject(vbo);
	cudaGLMapBufferObject( (void**)&dptr, vbo);

	//hipMemcpy(dptr,vert_data,9*sizeof(float),hipMemcpyHostToDevice);
	dim3 block(8,8,1);
	dim3 grid(256/block.x, 256/block.y, 1);
	fill_particles_kernel<<<1,1>>>(dptr, 256, 256, 1.0f );
	cudaGLUnmapBufferObject(vbo);
	std::cout << "VBO: " << vbo << std::endl;
}



void cuda_unreg_vbo(GLuint vbo)
{

    cudaGLUnregisterBufferObject(vbo);
    vbo = 0;
}

__global__ void fill_particles_kernel(float4 *pos, unsigned int width, unsigned int height, float time){


	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	    // calculate uv coordinates
	    float u = x / (float) width;
	    float v = y / (float) height;
	    u = u*2.0f - 1.0f;
	    v = v*2.0f - 1.0f;

	    // calculate simple sine wave pattern
	    float freq = 4.0f;
	    float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;
//
//	    pos[y*width+x].x = u;
//	    pos[y*width+x].y = w;
//	    pos[y*width+x].z = v;
//	    pos[y*width+x].w = 1.0f;

	pos[0].x = 0.0f;
	pos[0].y = 1.0f;
	pos[0].z = 0.0f;
	pos[0].w = 1.0f;

	pos[1].x = -1.0f;
	pos[1].y = -1.0f;
	pos[1].z = 0.0f;
	pos[1].w = 1.0f;

	pos[2].x = 1.0f;
	pos[2].y = -1.0f;
	pos[2].z = 0.0f;
	pos[2].w = 1.0f;

//		pos[0] = 0.0f;
//		pos[1] = 0.0f;
//		pos[2] = 0.0f;
//
//		pos[3] = 0.0f;
//		pos[4] = 0.0f;
//		pos[5] = 0.0f;
//
//		pos[6] = 0.0f;
//		pos[7] = 0.0f;
//		pos[8]= 0.0f;

	printf("hello from kernel\n");

}








__global__ void kernel(float4* pos, float * pdata, unsigned int width,
unsigned int height, int max_age, float time,
	 float randy1, float randy2, float randy3,
	 float tx, float ty, float tz)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned int arrayLoc = y*width*4 + x*4;
    unsigned int posLoc = y*width+x;

    float rx, ry, rz;
    float vx, vy, vz;
float dx, dy, dz, sum;

    // swarms dont have an age - can tweak this later
    // should initialize in main program
    // for gravity we could initially randomly scatter the particles
       // and let them coallese
       // similar to random swarm placement but no initial V for gravity
       // for gravity need to compute affect of all other particles
       // should try to do an M x M problem and see how fast it is
    // should also try a spring mesh - only need to talk to your neighbours

    if (pdata[arrayLoc] >= max_age)
       {
       rx = (randy1 - 0.5);
       ry = (randy2 - 0.5);
       rz = (randy3 - 0.5);

       pdata[arrayLoc] = 0; // age
       pdata[arrayLoc+1] = 0.001 * rx * rx * rx;
       pdata[arrayLoc+2] = 0.001 * ry * ry * ry;
       pdata[arrayLoc+3] = 0.001 * rz * rz * rz;

       // any new ones spawn near the target

// this gives some interesting effects
//tx = ty = tz = 0.0;
//pdata[arrayLoc+2] = 0.0;

       pos[posLoc].x = tx + 2.0 * rx * rx * rx;
       pos[posLoc].y = ty + 2.0 * ry * ry * ry;
       pos[posLoc].z = tz + 2.0 * rz * rz * rz;
       }

     pdata[arrayLoc] += 1;	    // increase age


     dx = (tx - pos[posLoc].x);
     dy = (ty - pos[posLoc].y);
     dz = (tz - pos[posLoc].z);
     sum = sqrt(dx*dx + dy*dy + dz*dz);

    // update the velocity
// I should give them a constant velocity but use this to update direction
    vx = 0.000005 * dx/sum;
    vy = 0.000005 * dy/sum;
    vz = 0.000005 * dz/sum;

    pdata[arrayLoc+1] = pdata[arrayLoc+1] + vx;
    pdata[arrayLoc+2] = pdata[arrayLoc+2] + vy;
    pdata[arrayLoc+3] = pdata[arrayLoc+3] + vz;

    float newX = pos[posLoc].x + pdata[arrayLoc+1];
    float newY = pos[posLoc].y + pdata[arrayLoc+2];
    float newZ = pos[posLoc].z + pdata[arrayLoc+3];


     // now need to modify the color info in the array

     pos[width*height + posLoc].x = 1.0;
     pos[width*height + posLoc].y = 1.0 - 0.5 * pdata[arrayLoc]/max_age;
     pos[width*height + posLoc].z = 1.0 - pdata[arrayLoc]/max_age;

    // write output vertex
     pos[posLoc] = make_float4(newX, newY, newZ, 1.0f);
}















